#include "hip/hip_runtime.h"
// AutoDecoderTrainerCUDA.cu
#include "AutoDecoderTrainerCUDA.h"
#include <hip/hip_runtime.h>
#include <algorithm>
#include <numeric>
#include <random>
#include <cstring>
#include <cstdio>

#ifndef CUDA_CHECK
#define CUDA_CHECK(expr) do { \
    hipError_t _err = (expr); \
    if (_err != hipSuccess) { \
        fprintf(stderr, "[CUDA] %s failed at %s:%d : %s\n", #expr, __FILE__, __LINE__, hipGetErrorString(_err)); \
        abort(); \
    } \
} while(0)
#endif

// ---------------- Device kernel ----------------
__device__ inline float dtanhf_out(float y) { return 1.f - y*y; }

extern "C" __global__
void adt_persistent_epoch(
    float* __restrict__ Wflat, float* __restrict__ Bflat,
    const int* __restrict__ layerIn, const int* __restrict__ layerOut, int numLayers,
    const int* __restrict__ wOffsets, const int* __restrict__ bOffsets,
    const float* __restrict__ Z_in, int numShapes, int latentDim,
    const float* __restrict__ coords, const float* __restrict__ targets, const int32_t* __restrict__ shapeIdx,
    const int32_t* __restrict__ order, int N, int coordDim,
    float lrW, float lrZ, float lambda,
    float* __restrict__ scratch,
    float* __restrict__ lossOut, float* __restrict__ lastOut
){
    const int tid = threadIdx.x;
    const int T   = blockDim.x;

    // compute max width from layerOut and layerIn[0]
    int maxWidth = 0;
    for (int l=0; l<numLayers; ++l) { maxWidth = max(maxWidth, layerOut[l]); }
    maxWidth = max(maxWidth, layerIn[0]);

    // scratch layout
    float* act0   = scratch;                  // len >= maxWidth
    float* act1   = act0 + maxWidth;          // len >= maxWidth
    float* delta  = act1 + maxWidth;          // len >= maxWidth
    float* dInput = delta + maxWidth;         // len >= layerIn[0]
    float* aStore = dInput + layerIn[0];      // (numLayers+1) * maxWidth
    auto A = [&](int l)->float* { return aStore + (size_t)l * (size_t)maxWidth; };

    auto matvec_par = [&](const float* W, const float* x, const float* b, float* y, int rows, int cols){
        for (int r = tid; r < rows; r += T) {
            float acc = b ? b[r] : 0.f;
            const float* wrow = W + (size_t)r * (size_t)cols;
            for (int c=0;c<cols;++c) acc += wrow[c]*x[c];
            y[r] = acc;
        }
        __syncthreads();
    };
    auto vec_tanh_par = [&](float* a, int n){
        for (int i=tid;i<n;i+=T) a[i] = tanhf(a[i]);
        __syncthreads();
    };
    auto sgd_rank1_par = [&](float* W, float* B, const float* u, const float* v, float alpha, int rows, int cols){
        for (int r=0;r<rows;++r) {
            float ar = alpha * u[r];
            for (int c=tid; c<cols; c+=T) {
                W[(size_t)r*(size_t)cols + (size_t)c] += ar * v[c];
            }
        }
        for (int r=tid; r<rows; r+=T) B[r] += alpha * u[r];
        __syncthreads();
    };
    auto WT_times_vec_par = [&](const float* W, const float* u, float* out, int rows, int cols){
        for (int c=tid; c<cols; c+=T) {
            float acc=0.f;
            for (int r=0;r<rows;++r) acc += W[(size_t)r*(size_t)cols + (size_t)c] * u[r];
            out[c] = acc;
        }
        __syncthreads();
    };
    auto hadamard_dtanh_par = [&](float* v, const float* a, int n){
        for (int i=tid;i<n;i+=T) v[i] *= dtanhf_out(a[i]);
        __syncthreads();
    };

    float totalLoss = 0.f;
    float last = 0.f;
    float* Z = const_cast<float*>(Z_in); // we update latents in-place

    for (int it=0; it<N; ++it) {
        const int idx = order[it];
        const int s   = shapeIdx[idx];
        const float* coord = coords + (size_t)idx * (size_t)coordDim;
        const float target = targets[idx];

        // input activation A(0) = [Z[s,*], coord]
        if (tid == 0) {
            for (int i=0;i<latentDim;++i) A(0)[i] = Z[(size_t)s*(size_t)latentDim + (size_t)i];
            for (int i=0;i<coordDim; ++i) A(0)[latentDim + i] = coord[i];
        }
        __syncthreads();

        // forward store
        float* cur = A(0);
        for (int l=0; l<numLayers; ++l) {
            const int rows = layerOut[l];
            const int cols = layerIn[l];
            float* W = Wflat + wOffsets[l];
            float* B = Bflat + bOffsets[l];

            matvec_par(W, cur, B, act1, rows, cols);
            if (l < numLayers-1) vec_tanh_par(act1, rows);

            for (int i=tid;i<rows;i+=T) A(l+1)[i] = act1[i];
            __syncthreads();
            cur = A(l+1);
        }

        const float pred = A(numLayers)[0];
        const float diff = pred - target;
        float sampleLoss = diff*diff;
        if (tid == 0) delta[0] = 2.f*diff;
        __syncthreads();

        // backward + immediate SGD
        for (int l=numLayers-1; l>=0; --l) {
            const int rows = layerOut[l];
            const int cols = layerIn[l];
            float* W = Wflat + wOffsets[l];
            float* B = Bflat + bOffsets[l];
            float* aPrev = A(l);

            // W,B update
            sgd_rank1_par(W, B, delta, aPrev, -lrW, rows, cols);

            // dInput for latent update (first layer)
            if (l == 0) {
                WT_times_vec_par(W, delta, dInput, rows, cols);
            }

            if (l > 0) {
                WT_times_vec_par(W, delta, act0, rows, cols);
                hadamard_dtanh_par(act0, aPrev, cols);
                for (int i=tid;i<cols;i+=T) delta[i] = act0[i];
                __syncthreads();
            }
        }

        // update latent
        for (int j=tid; j<latentDim; j+=T) {
            float zj = Z[(size_t)s*(size_t)latentDim + (size_t)j];
            float g  = dInput[j] + 2.f*lambda*zj;
            zj      -= lrZ * g;
            Z[(size_t)s*(size_t)latentDim + (size_t)j] = zj;
        }
        __syncthreads();

        if (tid == 0) {
            float lp=0.f;
            for (int j=0;j<latentDim;++j) { float zj = Z[(size_t)s*(size_t)latentDim + (size_t)j]; lp += lambda*(zj*zj); }
            totalLoss += sampleLoss + lp;
            last = sampleLoss + lp;
        }
        __syncthreads();
    }

    if (tid == 0) {
        *lossOut = totalLoss / (float)N;
        *lastOut = last;
    }
}

// ---------------- Host implementation ----------------
AutoDecoderTrainerCUDA::AutoDecoderTrainerCUDA() {}
AutoDecoderTrainerCUDA::~AutoDecoderTrainerCUDA(){ freeDevice_(); }

void AutoDecoderTrainerCUDA::setNetwork(const SimpleMLP& mlp, int numShapes, int latentDim, int coordDim){
    hostMLP_ = mlp;
    numShapes_ = numShapes;
    latentDim_ = latentDim;
    coordDim_  = coordDim;

    // init host latents
    std::mt19937 rng(42);
    std::normal_distribution<float> N01(0.f, 0.01f);
    h_Z_.assign((size_t)numShapes_ * (size_t)latentDim_, 0.f);
    for (auto& v : h_Z_) v = N01(rng);
}

void AutoDecoderTrainerCUDA::setSamples(const std::vector<ADTSample>& samples){
    dataset_ = samples;
    order_.resize((int)samples.size());
    std::iota(order_.begin(), order_.end(), 0);
}

void AutoDecoderTrainerCUDA::allocDevice_(){
    freeDevice_();

    // layer dims
    numLayers_ = (int)hostMLP_.hidden.size() + 1;
    h_layerIn_.clear(); h_layerOut_.clear();
    int inDim = hostMLP_.inputDim;
    for (size_t i=0;i<hostMLP_.hidden.size();++i){
        h_layerIn_.push_back(inDim);
        h_layerOut_.push_back(hostMLP_.hidden[i]);
        inDim = hostMLP_.hidden[i];
    }
    h_layerIn_.push_back(inDim);
    h_layerOut_.push_back(hostMLP_.outputDim);

    // offsets
    h_wOffsets_.assign(numLayers_+1, 0);
    h_bOffsets_.assign(numLayers_+1, 0);
    for (int l=0; l<numLayers_; ++l){
        h_wOffsets_[l+1] = h_wOffsets_[l] + h_layerOut_[l]*h_layerIn_[l];
        h_bOffsets_[l+1] = h_bOffsets_[l] + h_layerOut_[l];
    }

    // device allocs
    CUDA_CHECK(hipMalloc(&d_layerIn_,  sizeof(int)*numLayers_));
    CUDA_CHECK(hipMalloc(&d_layerOut_, sizeof(int)*numLayers_));
    CUDA_CHECK(hipMemcpy(d_layerIn_,  h_layerIn_.data(),  sizeof(int)*numLayers_, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_layerOut_, h_layerOut_.data(), sizeof(int)*numLayers_, hipMemcpyHostToDevice));

    CUDA_CHECK(hipMalloc(&d_wOffsets_, sizeof(int)*(numLayers_+1)));
    CUDA_CHECK(hipMalloc(&d_bOffsets_, sizeof(int)*(numLayers_+1)));
    CUDA_CHECK(hipMemcpy(d_wOffsets_, h_wOffsets_.data(), sizeof(int)*(numLayers_+1), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_bOffsets_, h_bOffsets_.data(), sizeof(int)*(numLayers_+1), hipMemcpyHostToDevice));

    int wCount = h_wOffsets_.back();
    int bCount = h_bOffsets_.back();
    CUDA_CHECK(hipMalloc(&d_W_, sizeof(float)*wCount));
    CUDA_CHECK(hipMalloc(&d_B_, sizeof(float)*bCount));

    int N = (int)dataset_.size();
    CUDA_CHECK(hipMalloc(&d_Z_,        sizeof(float)*(size_t)numShapes_*(size_t)latentDim_));
    CUDA_CHECK(hipMalloc(&d_coords_,   sizeof(float)*(size_t)N*(size_t)coordDim_));
    CUDA_CHECK(hipMalloc(&d_targets_,  sizeof(float)*(size_t)N));
    CUDA_CHECK(hipMalloc(&d_shapeIdx_, sizeof(int32_t)*(size_t)N));
    CUDA_CHECK(hipMalloc(&d_order_,    sizeof(int32_t)*(size_t)N));
}

void AutoDecoderTrainerCUDA::freeDevice_(){
    auto freeIf = [](auto*& p){ if (p) { hipFree(p); p=nullptr; } };
    freeIf(d_W_);
    freeIf(d_B_);
    freeIf(d_layerIn_);
    freeIf(d_layerOut_);
    freeIf(d_wOffsets_);
    freeIf(d_bOffsets_);
    freeIf(d_Z_);
    freeIf(d_coords_);
    freeIf(d_targets_);
    freeIf(d_shapeIdx_);
    freeIf(d_order_);
}

void AutoDecoderTrainerCUDA::uploadModel_(){
    // flatten upload
    int inDim = hostMLP_.inputDim;
    int oW=0, oB=0;
    for (size_t l=0; l<hostMLP_.hidden.size(); ++l){
        int rows = hostMLP_.hidden[l];
        int cols = inDim;
        CUDA_CHECK(hipMemcpy(d_W_ + oW, hostMLP_.weights[l].data(), sizeof(float)*rows*cols, hipMemcpyHostToDevice));
        CUDA_CHECK(hipMemcpy(d_B_ + oB, hostMLP_.biases[l].data(),  sizeof(float)*rows,     hipMemcpyHostToDevice));
        oW += rows*cols; oB += rows; inDim = rows;
    }
    // output
    {
        int rows = hostMLP_.outputDim;
        int cols = inDim;
        CUDA_CHECK(hipMemcpy(d_W_ + oW, hostMLP_.weights.back().data(), sizeof(float)*rows*cols, hipMemcpyHostToDevice));
        CUDA_CHECK(hipMemcpy(d_B_ + oB, hostMLP_.biases.back().data(),  sizeof(float)*rows,     hipMemcpyHostToDevice));
    }
    CUDA_CHECK(hipMemcpy(d_Z_, h_Z_.data(), sizeof(float)*h_Z_.size(), hipMemcpyHostToDevice));
}

void AutoDecoderTrainerCUDA::downloadModel_(){
    int inDim = hostMLP_.inputDim;
    int oW=0, oB=0;
    for (size_t l=0; l<hostMLP_.hidden.size(); ++l){
        int rows = hostMLP_.hidden[l];
        int cols = inDim;
        CUDA_CHECK(hipMemcpy(hostMLP_.weights[l].data(), d_W_ + oW, sizeof(float)*rows*cols, hipMemcpyDeviceToHost));
        CUDA_CHECK(hipMemcpy(hostMLP_.biases[l].data(),  d_B_ + oB, sizeof(float)*rows,     hipMemcpyDeviceToHost));
        oW += rows*cols; oB += rows; inDim = rows;
    }
    {
        int rows = hostMLP_.outputDim;
        int cols = inDim;
        CUDA_CHECK(hipMemcpy(hostMLP_.weights.back().data(), d_W_ + oW, sizeof(float)*rows*cols, hipMemcpyDeviceToHost));
        CUDA_CHECK(hipMemcpy(hostMLP_.biases.back().data(),  d_B_ + oB, sizeof(float)*rows,     hipMemcpyDeviceToHost));
    }
    CUDA_CHECK(hipMemcpy(h_Z_.data(), d_Z_, sizeof(float)*h_Z_.size(), hipMemcpyDeviceToHost));
}

void AutoDecoderTrainerCUDA::uploadData_(){
    const int N = (int)dataset_.size();
    std::vector<float> hcoords((size_t)N*(size_t)coordDim_);
    std::vector<float> htargets((size_t)N);
    std::vector<int32_t> hshape((size_t)N);
    for (int i=0;i<N;++i){
        std::memcpy(&hcoords[(size_t)i*(size_t)coordDim_], dataset_[i].coord.data(), sizeof(float)*(size_t)coordDim_);
        htargets[(size_t)i] = dataset_[i].target;
        hshape[(size_t)i]   = dataset_[i].shapeIndex;
    }
    CUDA_CHECK(hipMemcpy(d_coords_,  hcoords.data(),  sizeof(float)*hcoords.size(), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_targets_, htargets.data(), sizeof(float)*N,             hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_shapeIdx_,hshape.data(),   sizeof(int32_t)*N,           hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_order_,   order_.data(),   sizeof(int32_t)*N,           hipMemcpyHostToDevice));
}

ADTStats AutoDecoderTrainerCUDA::train(const ADTConfig& cfg){
    if (dataset_.empty()) throw std::runtime_error("No dataset set.");
    if (latentDim_ + coordDim_ != hostMLP_.inputDim)
        throw std::runtime_error("MLP inputDim must equal latentDim + coordDim");

    allocDevice_();
    uploadModel_();
    uploadData_();

    const int N = (int)dataset_.size();
    std::mt19937_64 rng(cfg.shuffleSeed);

    // Scratch size (in floats): 2*maxW + maxW + in0 + (L+1)*maxW
    int maxWidth = 0;
    for (int l=0;l<numLayers_;++l) maxWidth = std::max(maxWidth, h_layerOut_[l]);
    maxWidth = std::max(maxWidth, h_layerIn_[0]);
    size_t scratchFloats = (size_t)(4*maxWidth + (numLayers_+1)*maxWidth);

    float* d_scratch=nullptr;
    CUDA_CHECK(hipMalloc(&d_scratch, sizeof(float)*scratchFloats));
    float *d_loss=nullptr, *d_last=nullptr;
    CUDA_CHECK(hipMalloc(&d_loss, sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_last, sizeof(float)));

    ADTStats stats{};

    // choose threads
    const int threads = 128;

    for (int e=0; e<cfg.epochs; ++e){
        std::shuffle(order_.begin(), order_.end(), rng);
        CUDA_CHECK(hipMemcpy(d_order_, order_.data(), sizeof(int32_t)*N, hipMemcpyHostToDevice));

        adt_persistent_epoch<<<1, threads>>>(
            d_W_, d_B_,
            d_layerIn_, d_layerOut_, numLayers_,
            d_wOffsets_, d_bOffsets_,
            d_Z_, numShapes_, latentDim_, 
            d_coords_, d_targets_, d_shapeIdx_,
            d_order_, N, coordDim_,
            cfg.lrW, cfg.lrZ, cfg.lambda,
            d_scratch,
            d_loss, d_last
        );
        CUDA_CHECK(hipDeviceSynchronize());
        CUDA_CHECK(hipMemcpy(&stats.avgLoss, d_loss, sizeof(float), hipMemcpyDeviceToHost));
        CUDA_CHECK(hipMemcpy(&stats.lastLoss, d_last, sizeof(float), hipMemcpyDeviceToHost));
        stats.epochsRun++;
    }

    CUDA_CHECK(hipFree(d_scratch));
    CUDA_CHECK(hipFree(d_loss));
    CUDA_CHECK(hipFree(d_last));

    downloadModel_();
    freeDevice_();
    return stats;
}
